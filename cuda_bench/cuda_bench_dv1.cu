#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <sys/stat.h>
#include <time.h>
#include <chrono>
#include <thread>

#define __fq1 __device__
#define __gq1 __device__
#include "../src/madras_tt_dv1.hpp"

#include <hip/hip_runtime.h>

typedef struct {
  uint32_t key_loc;
  uint32_t key_len;
  union {
    uint32_t leaf_id;
    uint32_t leaf_seq;
  };
} key_ctx;

double time_taken_in_secs(clock_t t) {
  t = clock() - t;
  return ((double)t)/CLOCKS_PER_SEC;
}

clock_t print_time_taken(clock_t t, const char *msg) {
  double time_taken = time_taken_in_secs(t); // in seconds
  printf("%s %lf\n", msg, time_taken);
  return clock();
}

// Device-compatible madras_cuda_wrapper class
class madras_cuda_wrapper {
  public:
    madras_dv1::static_trie *st;
    key_ctx *lines;
    uint8_t *file_buf_lines;
    size_t file_size;
    size_t line_count;
    uint8_t *query_status;
    __device__ madras_cuda_wrapper() {}
    __device__ void init(uint8_t *_file_buf_lines, size_t _file_size, 
                         uint8_t *_file_buf_mdx, size_t _mdx_file_size,
                         key_ctx *_lines, size_t _line_count, uint8_t *_q_status) {
      lines = _lines;
      line_count = _line_count;
      query_status = _q_status;
      file_buf_lines = _file_buf_lines;
      file_size = _file_size;
      st = new madras_dv1::static_trie();
      st->load_static_trie(_file_buf_mdx);
      memset(query_status, '\0', line_count);
    }
    __device__ madras_dv1::static_trie *get_trie_inst() const {
      return st;
    }
};

// Kernel for initializing madras_cuda_wrapper on GPU
__global__ void init_madras_cuda_wrapper(madras_cuda_wrapper *d_nl, uint8_t *_file_buf_lines, size_t _file_size,
                    uint8_t *_file_buf_mdx, size_t _mdx_file_size,
                    key_ctx *_lines, size_t _line_count, uint8_t *_q_status) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
      d_nl->init(_file_buf_lines, _file_size, _file_buf_mdx, _mdx_file_size, _lines, _line_count, _q_status);
    }
}

// Kernel for invoking the lookup on the GPU
__global__ void lookup_kernel(madras_cuda_wrapper *d_cw, uint32_t start_idx, uint32_t num_queries) {
  uint32_t cuda_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (cuda_idx < num_queries) {
    uint32_t idx = start_idx + cuda_idx;
    madras_dv1::input_ctx in_ctx;
    key_ctx *ctx = &d_cw->lines[idx];
    in_ctx.key = d_cw->file_buf_lines + ctx->key_loc;
    in_ctx.key_len = ctx->key_len;
    bool is_success = d_cw->get_trie_inst()->lookup(in_ctx);
    ctx->leaf_id = d_cw->get_trie_inst()->leaf_rank1(in_ctx.node_id);
    uint8_t key_buf[256];
    size_t out_key_len;
    d_cw->get_trie_inst()->reverse_lookup(ctx->leaf_id, &out_key_len, key_buf);
    if (out_key_len == in_ctx.key_len && madras_dv1::cmn::memcmp(in_ctx.key, key_buf, out_key_len) == 0)
      d_cw->query_status[idx] = 1;
    // printf("Is success: %d\n", is_success);
    // printf("Node id: %u\n", in_ctx.node_id);
  }
}

void checkCudaError(hipError_t result, const char* msg) {
    if (result != hipSuccess) {
        printf("%s Error: %s\n", msg, hipGetErrorString(result));
        exit(EXIT_FAILURE);
    }
}

int main(int argc, const char *argv[]) {

  struct stat file_stat;
  memset(&file_stat, '\0', sizeof(file_stat));
  stat(argv[1], &file_stat);
  size_t file_size = file_stat.st_size;
  printf("File_name: %s, size: %lu\n", argv[1], file_size);

  FILE *fp = fopen(argv[1], "rb");
  if (fp == NULL) {
    perror("Could not open file; ");
    return 1;
  }
  uint8_t *file_buf = new uint8_t[file_size + 1];
  size_t res = fread(file_buf, 1, file_size, fp);
  if (res != file_size) {
    perror("Error reading file: ");
    free(file_buf);
    return 1;
  }
  fclose(fp);

  std::vector<key_ctx> lines;
  size_t line_count = 0;
  bool is_sorted = true;
  const uint8_t *prev_line = (const uint8_t *) "";
  size_t prev_line_len = 0;
  size_t line_len = 0;
  uint8_t *line = gen::extract_line(file_buf, line_len, file_size);
  do {
    if (gen::compare(line, line_len, prev_line, prev_line_len) != 0) {
      uint8_t *key = line;
      int key_len = line_len;
      if (gen::compare(key, key_len, prev_line, gen::min(prev_line_len, key_len)) < 0)
        is_sorted = false;
      lines.push_back((key_ctx) {(uint32_t) (line - file_buf), (uint32_t) line_len, UINT32_MAX});
      prev_line = line;
      prev_line_len = line_len;
      line_count++;
      if ((line_count % 100000) == 0) {
        printf(".");
        fflush(stdout);
      }
    }
    line = gen::extract_line(line, line_len, file_size - (line - file_buf) - line_len);
  } while (line != NULL);
  printf("\n");
  printf("Sorted? : %d\n", is_sorted);

  uint8_t *d_file_buf;
  hipMalloc(&d_file_buf, file_stat.st_size + 1);
  hipMemcpy(d_file_buf, file_buf, file_stat.st_size + 1, hipMemcpyHostToDevice);

  char mdx_file_name[strlen(argv[1]) + 5];
  strcpy(mdx_file_name, argv[1]);
  strcat(mdx_file_name, ".mdx");
  memset(&file_stat, '\0', sizeof(file_stat));
  stat(mdx_file_name, &file_stat);
  size_t mdx_file_size = file_stat.st_size;
  printf("MDX File_name: %s, size: %lu\n", mdx_file_name, mdx_file_size);
  fp = fopen(mdx_file_name, "rb");
  if (fp == NULL) {
    perror("Could not open mdx file; ");
    free(file_buf);
    return 1;
  }
  uint8_t *mdx_file_buf = new uint8_t[mdx_file_size + 1];
  res = fread(mdx_file_buf, 1, mdx_file_size, fp);
  if (res != mdx_file_size) {
    perror("Error reading mdx file: ");
    free(file_buf);
    free(mdx_file_buf);
    return 1;
  }
  fclose(fp);

  uint8_t *d_file_buf_lines;
  hipMalloc(&d_file_buf_lines, file_size + 1);
  hipMemcpy(d_file_buf_lines, file_buf, file_size + 1, hipMemcpyHostToDevice);

  uint8_t *d_file_buf_mdx;
  hipMallocManaged(&d_file_buf_mdx, mdx_file_size + 1);
  hipMemcpy(d_file_buf_mdx, mdx_file_buf, mdx_file_size + 1, hipMemcpyHostToDevice);
  // uint8_t *h_file_buf_mdx;     // Host pointer
  // uint8_t *d_file_buf_mdx;     // Device pointer
  // hipHostAlloc((void**)&h_file_buf_mdx, mdx_file_size + 1, hipHostMallocMapped);
  // memcpy(h_file_buf_mdx, mdx_file_buf, mdx_file_size + 1);
  // hipHostGetDevicePointer(&d_file_buf_mdx, h_file_buf_mdx, 0);

  key_ctx *d_lines;
  hipMalloc(&d_lines, sizeof(key_ctx) * lines.size());
  hipMemcpy(d_lines, lines.data(), sizeof(key_ctx) * lines.size(), hipMemcpyHostToDevice);

  uint8_t *d_query_status;
  hipMalloc(&d_query_status, lines.size());

  madras_cuda_wrapper *d_cw;
  hipMalloc(&d_cw, sizeof(madras_cuda_wrapper));

  //hipDeviceSetLimit(hipLimitStackSize, 4096);

  // Initialize the `madras_cuda_wrapper` object with GPU data
  init_madras_cuda_wrapper<<<1, 1>>>(d_cw, d_file_buf_lines, file_size + 1,
        d_file_buf_mdx, mdx_file_size + 1, d_lines, lines.size(), d_query_status);
  hipDeviceSynchronize(); // Ensure initialization completes before lookup_kernel

  clock_t t = clock();

  // Launch the kernel to perform lookups
  size_t threads_per_block = 768;
  size_t blocks = 12;
  size_t capacity = blocks * threads_per_block;
  size_t iter_count = lines.size() / capacity;
  if ((lines.size() % capacity) > 0)
    iter_count++;
  for (size_t i = 0; i < iter_count; i++) {
    size_t query_count = capacity;
    if (i == (iter_count - 1) && (lines.size() % capacity) > 0)
      query_count = (lines.size() % capacity);
    lookup_kernel<<<blocks, threads_per_block>>>(d_cw, i * capacity, query_count);
    hipDeviceSynchronize();
  }

  t = print_time_taken(t, "Time taken for retrieve: ");

  // Copy results back to host
  uint8_t *query_status = new uint8_t[lines.size()];
  hipMemcpy(query_status, d_query_status, lines.size(), hipMemcpyDeviceToHost);
  // hipMemcpy(lines.data(), d_lines, sizeof(key_ctx) * lines.size(), hipMemcpyDeviceToHost);

  size_t success_count = 0;
  for (size_t i = 0; i < lines.size(); i++) {
    if (query_status[i] == 1)
      success_count++;
  }
  printf("Success count: %lu, Total: %lu\n", success_count, lines.size());

  // Cleanup GPU memory
  hipFree(d_cw);
  hipFree(d_lines);
  hipFree(d_file_buf);
  hipFree(d_file_buf_mdx);
  hipFree(d_query_status);

  delete [] file_buf;
  delete [] mdx_file_buf;
  delete [] query_status;

  return 0;

}
