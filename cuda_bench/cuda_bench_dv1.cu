#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <sys/stat.h>
#include <time.h>

#define __fq1 __device__
#define __gq1 __device__
#include "../src/madras_tt_dv1.hpp"

#include <hip/hip_runtime.h>

typedef struct {
  uint32_t key_loc;
  uint32_t key_len;
  union {
    uint32_t leaf_id;
    uint32_t leaf_seq;
  };
} key_ctx;

double time_taken_in_secs(clock_t t) {
  t = clock() - t;
  return ((double)t)/CLOCKS_PER_SEC;
}

clock_t print_time_taken(clock_t t, const char *msg) {
  double time_taken = time_taken_in_secs(t); // in seconds
  printf("%s %lf\n", msg, time_taken);
  return clock();
}

// Device-compatible madras_cuda_wrapper class
class madras_cuda_wrapper {
  public:
    madras_dv1::static_trie *st;
    key_ctx *lines;
    uint8_t *file_buf_lines;
    size_t file_size;
    size_t line_count;
    uint8_t *query_status;
    __device__ madras_cuda_wrapper() {}
    __device__ void init(uint8_t *_file_buf_lines, size_t _file_size, 
                         uint8_t *_file_buf_mdx, size_t _mdx_file_size,
                         key_ctx *_lines, size_t _line_count, uint8_t *_q_status) {
      lines = _lines;
      line_count = _line_count;
      query_status = _q_status;
      file_buf_lines = _file_buf_lines;
      file_size = _file_size;
      st = new madras_dv1::static_trie();
      st->load_static_trie(_file_buf_mdx);
      memset(query_status, '\0', line_count);
    }
    __device__ madras_dv1::static_trie *get_trie_inst() const {
      return st;
    }
};

// Kernel for initializing madras_cuda_wrapper on GPU
__global__ void init_madras_cuda_wrapper(madras_cuda_wrapper *d_nl, uint8_t *_file_buf_lines, size_t _file_size,
                    uint8_t *_file_buf_mdx, size_t _mdx_file_size,
                    key_ctx *_lines, size_t _line_count, uint8_t *_q_status) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
      d_nl->init(_file_buf_lines, _file_size, _file_buf_mdx, _mdx_file_size, _lines, _line_count, _q_status);
    }
}

// Kernel for invoking the lookup on the GPU
__global__ void lookup_kernel(madras_cuda_wrapper *d_cw, size_t start_idx, size_t num_queries) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_queries) {
    madras_dv1::input_ctx in_ctx;
    in_ctx.key = d_cw->file_buf_lines + d_cw->lines[start_idx + idx].key_loc;
    in_ctx.key_len = d_cw->lines[start_idx + idx].key_len;
    bool is_success = d_cw->get_trie_inst()->lookup(in_ctx);
    d_cw->query_status[start_idx + idx] = is_success;
    // printf("Is success: %d\n", is_success);
    // printf("Node id: %u\n", in_ctx.node_id);
  }
}

int main(int argc, const char *argv[]) {

  clock_t t = clock();

  struct stat file_stat;
  memset(&file_stat, '\0', sizeof(file_stat));
  stat(argv[1], &file_stat);
  size_t file_size = file_stat.st_size;
  printf("File_name: %s, size: %lu\n", argv[1], file_size);

  FILE *fp = fopen(argv[1], "rb");
  if (fp == NULL) {
    perror("Could not open file; ");
    return 1;
  }
  uint8_t *file_buf = new uint8_t[file_size + 1];
  size_t res = fread(file_buf, 1, file_size, fp);
  if (res != file_size) {
    perror("Error reading file: ");
    free(file_buf);
    return 1;
  }
  fclose(fp);

  std::vector<key_ctx> lines;
  size_t line_count = 0;
  bool is_sorted = true;
  const uint8_t *prev_line = (const uint8_t *) "";
  size_t prev_line_len = 0;
  size_t line_len = 0;
  uint8_t *line = gen::extract_line(file_buf, line_len, file_size);
  do {
    if (gen::compare(line, line_len, prev_line, prev_line_len) != 0) {
      uint8_t *key = line;
      int key_len = line_len;
      if (gen::compare(key, key_len, prev_line, gen::min(prev_line_len, key_len)) < 0)
        is_sorted = false;
      lines.push_back((key_ctx) {(uint32_t) (line - file_buf), (uint32_t) line_len, 0});
      prev_line = line;
      prev_line_len = line_len;
      line_count++;
      if ((line_count % 100000) == 0) {
        printf(".");
        fflush(stdout);
      }
    }
    line = gen::extract_line(line, line_len, file_size - (line - file_buf) - line_len);
  } while (line != NULL);
  printf("\n");
  printf("Sorted? : %d\n", is_sorted);

  uint8_t *d_file_buf;
  hipMalloc(&d_file_buf, file_stat.st_size + 1);
  hipMemcpy(d_file_buf, file_buf, file_stat.st_size + 1, hipMemcpyHostToDevice);

  char mdx_file_name[strlen(argv[1]) + 5];
  strcpy(mdx_file_name, argv[1]);
  strcat(mdx_file_name, ".mdx");
  memset(&file_stat, '\0', sizeof(file_stat));
  stat(mdx_file_name, &file_stat);
  size_t mdx_file_size = file_stat.st_size;
  printf("MDX File_name: %s, size: %lu\n", mdx_file_name, mdx_file_size);
  fp = fopen(mdx_file_name, "rb");
  if (fp == NULL) {
    perror("Could not open mdx file; ");
    free(file_buf);
    return 1;
  }
  uint8_t *mdx_file_buf = new uint8_t[mdx_file_size + 1];
  res = fread(mdx_file_buf, 1, mdx_file_size, fp);
  if (res != mdx_file_size) {
    perror("Error reading mdx file: ");
    free(file_buf);
    free(mdx_file_buf);
    return 1;
  }
  fclose(fp);

  uint8_t *d_file_buf_lines;
  hipMalloc(&d_file_buf_lines, file_size + 1);
  hipMemcpy(d_file_buf_lines, file_buf, file_size + 1, hipMemcpyHostToDevice);

  uint8_t *d_file_buf_mdx;
  hipMalloc(&d_file_buf_mdx, mdx_file_size + 1);
  hipMemcpy(d_file_buf_mdx, mdx_file_buf, mdx_file_size + 1, hipMemcpyHostToDevice);

  key_ctx *d_lines;
  hipMalloc(&d_lines, sizeof(key_ctx) * lines.size());
  hipMemcpy(d_lines, lines.data(), sizeof(key_ctx) * lines.size(), hipMemcpyHostToDevice);

  uint8_t *d_query_status;
  hipMalloc(&d_query_status, lines.size());

  madras_cuda_wrapper *d_cw;
  hipMalloc(&d_cw, sizeof(madras_cuda_wrapper));

  //hipDeviceSetLimit(hipLimitStackSize, 4096);

  // Initialize the `madras_cuda_wrapper` object with GPU data
  init_madras_cuda_wrapper<<<1, 1>>>(d_cw, d_file_buf_lines, file_size + 1,
        d_file_buf_mdx, mdx_file_size + 1, d_lines, lines.size(), d_query_status);
  hipDeviceSynchronize(); // Ensure initialization completes before lookup_kernel

  // Launch the kernel to perform lookups
  size_t threads_per_block = 1024;
  size_t blocks = 6;
  size_t capacity = blocks * threads_per_block;
  size_t iter_count = lines.size() / capacity;
  if ((lines.size() % capacity) > 0)
    iter_count++;
  for (size_t i = 0; i < iter_count; i++) {
    size_t query_count = capacity;
    if (i == (iter_count - 1) && (lines.size() % capacity) > 0)
      query_count = (lines.size() % capacity);
    lookup_kernel<<<blocks, threads_per_block>>>(d_cw, i * capacity, query_count);
    hipDeviceSynchronize();
  }

  // lookup_kernel<<<2, 10>>>(d_cw, num_queries, d_query_status);

  // Copy results back to host
  uint8_t *query_status = new uint8_t[lines.size()];
  hipMemcpy(query_status, d_query_status, lines.size(), hipMemcpyDeviceToHost);

  t = print_time_taken(t, "Time taken for retrieve: ");

  size_t success_count = 0;
  for (size_t i = 0; i < lines.size(); i++) {
    if (query_status[i] == 1)
      success_count++;
  }
  printf("Success count: %lu, Total: %lu\n", success_count, lines.size());

  // Cleanup GPU memory
  hipFree(d_cw);
  hipFree(d_lines);
  hipFree(d_file_buf);
  hipFree(d_file_buf_mdx);
  hipFree(d_query_status);

  delete [] file_buf;
  delete [] mdx_file_buf;
  delete [] query_status;

  return 0;

}
