#include "hip/hip_runtime.h"
#include "common_mt_dv1.hpp"

#define __fq1 __device__
#define __gq1 __device__
#include "../src/madras_dv1.hpp"

#include <hip/hip_runtime.h>

// Device-compatible madras_cuda_wrapper class
class madras_cuda_wrapper {
  public:
    madras_dv1::static_trie *st;
    key_ctx *lines;
    uint8_t *file_buf_lines;
    size_t file_size;
    size_t line_count;
    uint8_t *query_status;
    __device__ madras_cuda_wrapper() {}
    __device__ void init(uint8_t *_file_buf_lines, size_t _file_size, 
                         uint8_t *_file_buf_mdx, size_t _mdx_file_size,
                         key_ctx *_lines, size_t _line_count, uint8_t *_q_status) {
      lines = _lines;
      line_count = _line_count;
      query_status = _q_status;
      file_buf_lines = _file_buf_lines;
      file_size = _file_size;
      st = new madras_dv1::static_trie();
      st->load_static_trie(_file_buf_mdx);
      memset(query_status, '\0', line_count);
    }
    __device__ madras_dv1::static_trie *get_trie_inst() const {
      return st;
    }
};

// Kernel for initializing madras_cuda_wrapper on GPU
__global__ void init_madras_cuda_wrapper(madras_cuda_wrapper *d_nl, uint8_t *_file_buf_lines, size_t _file_size,
                    uint8_t *_file_buf_mdx, size_t _mdx_file_size,
                    key_ctx *_lines, size_t _line_count, uint8_t *_q_status) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
      d_nl->init(_file_buf_lines, _file_size, _file_buf_mdx, _mdx_file_size, _lines, _line_count, _q_status);
    }
}

// Kernel for invoking the lookup on the GPU
__global__ void lookup_1x1_kernel(madras_cuda_wrapper *d_cw, uint32_t num_queries) {
  madras_dv1::input_ctx in_ctx;
  uint8_t key_buf[256];
  size_t out_key_len;
  for (uint32_t idx = 0; idx < num_queries; idx++) {
    key_ctx *ctx = &d_cw->lines[idx];
    in_ctx.key = d_cw->file_buf_lines + ctx->key_loc;
    in_ctx.key_len = ctx->key_len;
    bool is_success = d_cw->get_trie_inst()->lookup(in_ctx);
    ctx->leaf_id = d_cw->get_trie_inst()->leaf_rank1(in_ctx.node_id);
    d_cw->get_trie_inst()->reverse_lookup(ctx->leaf_id, &out_key_len, key_buf);
    if (out_key_len == in_ctx.key_len && madras_dv1::cmn::memcmp(in_ctx.key, key_buf, out_key_len) == 0)
      d_cw->query_status[idx] = is_success;
    // printf("Is success: %d\n", is_success);
    // printf("Node id: %u\n", in_ctx.node_id);
  }
}

// Kernel for invoking the lookup on the GPU
__global__ void lookup_kernel(madras_cuda_wrapper *d_cw, uint32_t start_idx, uint32_t num_queries) {
  uint32_t cuda_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (cuda_idx < num_queries) {
    uint32_t idx = start_idx + cuda_idx;
    madras_dv1::input_ctx in_ctx;
    key_ctx *ctx = &d_cw->lines[idx];
    in_ctx.key = d_cw->file_buf_lines + ctx->key_loc;
    in_ctx.key_len = ctx->key_len;
    bool is_success = d_cw->get_trie_inst()->lookup(in_ctx);
    ctx->leaf_id = d_cw->get_trie_inst()->leaf_rank1(in_ctx.node_id);
    uint8_t key_buf[256];
    size_t out_key_len;
    d_cw->get_trie_inst()->reverse_lookup(ctx->leaf_id, &out_key_len, key_buf);
    if (out_key_len == in_ctx.key_len && madras_dv1::cmn::memcmp(in_ctx.key, key_buf, out_key_len) == 0)
      d_cw->query_status[idx] = is_success;
    // printf("Is success: %d\n", is_success);
    // printf("Node id: %u\n", in_ctx.node_id);
  }
}

void checkCudaError(hipError_t result, const char* msg) {
    if (result != hipSuccess) {
        printf("%s Error: %s\n", msg, hipGetErrorString(result));
        exit(EXIT_FAILURE);
    }
}

int main(int argc, const char *argv[]) {

  if (argc < 4) {
    printf("Usage: cuda_bench <file_name> <num_threads> <num_blocks>\n");
    return 1;
  }

  hipSetDevice(0);
  hipSetDeviceFlags(hipDeviceScheduleYield);
  hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

  std::vector<key_ctx> lines;
  size_t file_size;
  uint8_t *file_buf = load_lines(argv[1], lines, file_size);

  uint8_t *d_file_buf;
  hipMalloc(&d_file_buf, file_size + 1);
  hipMemcpy(d_file_buf, file_buf, file_size + 1, hipMemcpyHostToDevice);

  size_t mdx_file_size;
  uint8_t *mdx_file_buf = load_mdx_file(argv[1], mdx_file_size);

  uint8_t *d_file_buf_lines;
  hipMalloc(&d_file_buf_lines, file_size + 1);
  hipMemcpy(d_file_buf_lines, file_buf, file_size + 1, hipMemcpyHostToDevice);

  uint8_t *d_file_buf_mdx;
  hipMallocManaged(&d_file_buf_mdx, mdx_file_size + 1);
  hipMemcpy(d_file_buf_mdx, mdx_file_buf, mdx_file_size + 1, hipMemcpyHostToDevice);
  // uint8_t *h_file_buf_mdx; // Host pointer
  // uint8_t *d_file_buf_mdx; // Device pointer
  // hipHostAlloc((void**)&h_file_buf_mdx, mdx_file_size + 1, hipHostMallocMapped);
  // memcpy(h_file_buf_mdx, mdx_file_buf, mdx_file_size + 1);
  // hipHostGetDevicePointer(&d_file_buf_mdx, h_file_buf_mdx, 0);

  key_ctx *d_lines;
  hipMalloc(&d_lines, sizeof(key_ctx) * lines.size());
  hipMemcpy(d_lines, lines.data(), sizeof(key_ctx) * lines.size(), hipMemcpyHostToDevice);

  uint8_t *d_query_status;
  hipMalloc(&d_query_status, lines.size());

  madras_cuda_wrapper *d_cw;
  hipMalloc(&d_cw, sizeof(madras_cuda_wrapper));

  //hipDeviceSetLimit(hipLimitStackSize, 4096);

  // Initialize the `madras_cuda_wrapper` object with GPU data
  init_madras_cuda_wrapper<<<1, 1>>>(d_cw, d_file_buf_lines, file_size + 1,
        d_file_buf_mdx, mdx_file_size + 1, d_lines, lines.size(), d_query_status);
  hipDeviceSynchronize(); // Ensure initialization completes before lookup_kernel

  struct timespec t;
  clock_gettime(CLOCK_REALTIME, &t);

  // Launch the kernel to perform lookups
  size_t threads_per_block = atoi(argv[2]);
  size_t blocks = atoi(argv[3]);
  size_t capacity = blocks * threads_per_block;
  size_t iter_count = lines.size() / capacity;
  if ((lines.size() % capacity) > 0)
    iter_count++;
  for (size_t i = 0; i < iter_count; i++) {
    size_t query_count = capacity;
    if (i == (iter_count - 1) && (lines.size() % capacity) > 0)
      query_count = (lines.size() % capacity);
    lookup_kernel<<<blocks, threads_per_block>>>(d_cw, i * capacity, query_count);
    hipDeviceSynchronize();
  }

  // lookup_1x1_kernel<<<1, 1>>>(d_cw, lines.size());
  // hipDeviceSynchronize();

  //hipDeviceSynchronize();

  t = print_time_taken(t, "Time taken for retrieve: ");

  // Copy results back to host
  uint8_t *query_status = new uint8_t[lines.size()];
  hipMemcpy(query_status, d_query_status, lines.size(), hipMemcpyDeviceToHost);
  // hipMemcpy(lines.data(), d_lines, sizeof(key_ctx) * lines.size(), hipMemcpyDeviceToHost);

  size_t success_count = 0;
  for (size_t i = 0; i < lines.size(); i++) {
    if (query_status[i] == 1)
      success_count++;
  }
  printf("Success count: %lu, Total: %lu\n", success_count, lines.size());

  // Cleanup GPU memory
  hipFree(d_cw);
  hipFree(d_lines);
  hipFree(d_file_buf);
  hipFree(d_file_buf_mdx);
  hipFree(d_query_status);

  delete [] file_buf;
  delete [] mdx_file_buf;
  delete [] query_status;

  return 0;

}
